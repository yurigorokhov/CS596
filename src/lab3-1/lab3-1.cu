#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 1 - Global vs Shared memory speeds
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/cuda_util.h"

#define ITERATIONS 10000000

__global__ void register_mem_kernel();
__global__ void shared_mem_kernel();

int main() {
	hipEvent_t start, stop;
	float elapsedTime;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Register Memory
	hipEventRecord(start,0);
	register_mem_kernel<<<1, 1>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("Register memory: %f\n", elapsedTime);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Shared Memory
	hipEventRecord(start,0);
	shared_mem_kernel<<<1, 1>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("Shared memory: %f\n", elapsedTime);

    return 0;
}

__global__ void register_mem_kernel() {
	int location;
	for(int i = 0; i < ITERATIONS; i++) {
		int tmp = location >> 1;
		location = tmp;
	}
}

__global__ void shared_mem_kernel() {
	__shared__ int location;
	for(int i = 0; i < ITERATIONS; i++) {
		int tmp = location >> 1;
		location = tmp;
	}
}
