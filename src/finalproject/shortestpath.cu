/**
 * Yuri Gorokhov
 * final project - Shortest path
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#include "stack.h"
#include "../include/cuda_util.h"

#define MAT_SIZE 16
#define MAX_ELEMENT 1024

// Matrix to be represented as single array
typedef struct {
  int * array;
  int columns;
} Matrix;

int shortest_path_cpu(Matrix*, Stack*);

int main() {
  
  // Create a matrix and populate it with random data
  Matrix mat;
  mat.array = (int*)malloc(MAT_SIZE * MAT_SIZE * sizeof(int));
  mat.columns = MAT_SIZE;
  srand ( time(NULL) );
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++) {
    mat.array[i] = rand() % MAX_ELEMENT;
  }
  
  // Create a result stack
  Stack result;
  stack_init(&result, MAT_SIZE*2);
  
  // Compute shortest path with cpu
  shortest_path_cpu(&mat, &result);
  
  // Print path taken 
  printf("\nShortest Path: ");
  while(!is_empty(&result)) {
    printf("%i, ", pop(&result));
  }
  
  return 0;
}

/*
 * Path is returned as 1 for right, 0 for down
 * @return - int shortest path
 */
int shortest_path_cpu(Matrix * mat, Stack *result) {
  
  // Create temp matrix to store the path sums
  Matrix sum_matrix;
  mat.columns = mat->columns;
  mat.array = (int*)malloc(mat->columns * mat->columns * sizeof(int)));
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++)
    mat.array[i] = 0;
  
  // Calculate shortest path
  for(int i = 0; i < mat->columns-1; i++) {
    
  }
}



