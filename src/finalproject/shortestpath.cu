/**
 * Yuri Gorokhov
 * final project - Shortest path
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#include "stack.h"
#include "../include/cuda_util.h"

#define MAT_SIZE 16
#define MAX_ELEMENT 1024

// Matrix to be represented as single array
typedef struct {
  int * array;
  int columns;
} Matrix;

int shortest_path_cpu(Matrix*, Stack*);

int main() {
  
  // Create a matrix and populate it with random data
  Matrix mat;
  mat.array = (int*)malloc(MAT_SIZE * MAT_SIZE * sizeof(int));
  mat.columns = MAT_SIZE;
  srand ( time(NULL) );
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++) {
    mat.array[i] = rand() % MAX_ELEMENT;
  }
  
  // Create a result stack
  Stack result;
  stack_init(&result, MAT_SIZE*2);
  
  // Compute shortest path with cpu
  int shortestpath = shortest_path_cpu(&mat, &result);
  
  // Print path taken 
  printf("\nShortest Path: %i -> ", shortestpath);
  while(!is_empty(&result)) {
    printf("%i,", pop(&result));
  }
  printf("\n");
  
  return 0;
}

/*
 * Path is returned as 1 for right, 0 for down
 * Note: this algorithm goes from top left to bottom right corner
 * @return - int shortest path
 */
int shortest_path_cpu(Matrix * mat, Stack *result) {
  
  // Create temp matrix to store the path sums
  Matrix sum_matrix;
  int cols = mat->columns;
  sum_matrix.columns = cols;
  sum_matrix.array = (int*)malloc(cols * cols * sizeof(int));
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++)
    sum_matrix.array[i] = 0;
  
  // Initialize corner-most element
  sum_matrix.array[0] = mat->array[0];
  
  // Calculate sum of first half
  for(int i = 1; i < cols; i++) {
    for(int j = i, k = 0; j >= 0; j--, k++) {
      
      // record the shortest path to current node
      if(j == 0) {
	sum_matrix.array[cols * k + j] = sum_matrix.array[cols * (k-1) + j] + mat->array[cols * k + j];
      } else if(k == 0) {
	sum_matrix.array[cols * k + j] = sum_matrix.array[cols * k + j-1] + mat->array[cols * k + j];
      } else {
	int path1 = sum_matrix.array[cols * (k-1) + j];
	int path2 = sum_matrix.array[cols * k + j-1];
	if(path1 <= path2) {
	  sum_matrix.array[cols * k + j] = path1 + mat->array[cols * k + j];
	} else {
	  sum_matrix.array[cols * k + j] = path2 + mat->array[cols * k + j];
	}
      }
    }
  }
  
   // Calculate sum of second half
  for(int i = 1; i < cols; i++) {
    for(int j = i, k = cols-1; j < cols; j++, k--) {
      
      // record the shortest path to current node
	int path1 = sum_matrix.array[cols * (k-1) + j];
	int path2 = sum_matrix.array[cols * k + j-1];
	if(path1 <= path2) {
	  sum_matrix.array[cols * k + j] = path1 + mat->array[cols * k + j];
	} else {
	  sum_matrix.array[cols * k + j] = path2 + mat->array[cols * k + j];
	}
    }
  }
  
  int shortestpath = sum_matrix.array[cols*cols-1];
  
  // Put shortest path onto stack
  int j = cols-1, k = cols-1;
  while(j > 0 || k > 0) {
    if(j == 0) {
      k--;
      push(result, 0);
    } else if(k == 0) {
      j--;
      push(result, 1);
    } else {
      int path1 = sum_matrix.array[cols * (k-1) + j];
      int path2 = sum_matrix.array[cols * k + j-1];
      if(path1 < path2) {
	k--;
	push(result, 0);
      } else {
	j--;
	push(result, 1);
      }
    }
  }
  
  free(sum_matrix.array);
  return shortestpath;
}



