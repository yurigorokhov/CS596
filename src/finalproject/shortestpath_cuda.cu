#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * final project - Shortest path
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#include "stack.h"
#include "../include/cuda_util.h"

#define MAT_SIZE 16
#define MAX_ELEMENT 1024

// Matrix to be represented as single array
typedef struct {
  int * array;
  int columns;
} Matrix;

__global__ void shortest_path_cuda(int*, int*);

int main() {
  
  // Create a matrix and populate it with random data
  Matrix mat;
  mat.array = (int*)malloc(MAT_SIZE * MAT_SIZE * sizeof(int));
  mat.columns = MAT_SIZE;
  srand ( time(NULL) );
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++) {
    mat.array[i] = rand() % MAX_ELEMENT;
  }
  
  // Create a result stack
  Stack result;
  stack_init(&result, MAT_SIZE*2);
  
  // Copy matrix to global memory
  int *DevMat, *dev_shortest_path;
  cudasafe( hipMalloc((void**)&dev_shortest_path, sizeof(int)), "hipMalloc" );
  cudasafe( hipMalloc((void**)&DevMat, sizeof(MAT_SIZE * MAT_SIZE * sizeof(int))), "hipMalloc" );
  cudasafe( hipMemcpy(DevMat, mat.array, sizeof(MAT_SIZE * MAT_SIZE * sizeof(int)) , hipMemcpyHostToDevice), "hipMemcpy" );
  
  // Compute shortest path with cpu
  int shortestpath = 0;
  shortest_path_cuda<<<1,MAT_SIZE>>>(DevMat, dev_shortest_path);
  cudasafe( hipMemcpy(&shortestpath, dev_shortest_path, sizeof(int), hipMemcpyDeviceToHost) ,"hipMemcpy");
  cudasafe( hipFree(DevMat), "hipFree" );
  cudasafe( hipFree(dev_shortest_path), "hipFree" );
  
  // Print path taken 
  printf("\nShortest Path: %i -> ", shortestpath);
  while(!is_empty(&result)) {
    printf("%i,", pop(&result));
  }
  printf("\n");
  
  return 0;
}

/*
 * Path is returned as 1 for right, 0 for down
 * Note: this algorithm goes from top left to bottom right corner
 */
__global__ void shortest_path_cuda(int * mat, int * shortestpath) {
  
  // Copy Mat to shared memory
  __shared__ int matrix[MAT_SIZE * MAT_SIZE];
  
  if(threadIdx.x == 0) {
    matrix[0] = mat[0];
  }
  
  // Compute shortest path part1
  for(int i = 1; i < MAT_SIZE; i++) {
    __syncthreads();
    
    // only use the threads we need
    if(threadIdx.x <= i) {
      int idx = threadIdx.x * MAT_SIZE + (i - threadIdx.x);
      if(threadIdx.x == 0) {
	matrix[idx] = matrix[idx-1] + mat[idx];
      } else if(i - threadIdx.x == 0) {
	matrix[idx] = matrix[idx - MAT_SIZE] + mat[idx];
      } else {	
	int path1 = idx - 1;
	int path2 = idx - MAT_SIZE;
	matrix[idx] = matrix[path1] < matrix[path2] 
		      ? matrix[path1] + mat[idx] 
		      : matrix[path2] + mat[idx];
      }
    }
  }
  
  // Compute shortest path part2
  for(int i = 1; i < MAT_SIZE; i++) {
    __syncthreads();
    
    // only use the threads we need
    if(threadIdx.x < MAT_SIZE - i) {
      int idx = (MAT_SIZE - threadIdx.x - 1) + (threadIdx.x + i) * MAT_SIZE;
      int path1 = idx - 1;
      int path2 = idx - MAT_SIZE;
      matrix[idx] = matrix[path1] < matrix[path2] 
		    ? matrix[path1] + mat[idx] 
		    : matrix[path2] + mat[idx];
    }
  } 
  
  if(threadIdx.x == 0)
    *shortestpath = matrix[MAT_SIZE * MAT_SIZE - 1];
  __syncthreads();
}



