#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * final project - Shortest path
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#include "stack.h"
#include "../include/cuda_util.h"

#define MAT_SIZE 16
#define MAX_ELEMENT 1024

// Matrix to be represented as single array
typedef struct {
  int * array;
  int columns;
} Matrix;

__global__ void shortest_path_cuda(int*);

int main() {
  
  // Create a matrix and populate it with random data
  Matrix mat;
  mat.array = (int*)malloc(MAT_SIZE * MAT_SIZE * sizeof(int));
  mat.columns = MAT_SIZE;
  srand ( time(NULL) );
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++) {
    mat.array[i] = rand() % MAX_ELEMENT;
  }
  
  // Create a result stack
  Stack result;
  stack_init(&result, MAT_SIZE*2);
  
  // Copy matrix to global memory
  int *DevMat;
  cudasafe( hipMalloc((void**)&DevMat, sizeof(MAT_SIZE * MAT_SIZE * sizeof(int))), "hipMalloc" );
  cudasafe( hipMemcpy(DevMat, mat.array, sizeof(MAT_SIZE * MAT_SIZE * sizeof(int)) , hipMemcpyHostToDevice), "hipMemcpy" );
  
  // Compute shortest path with cpu
  int shortestpath = 0;
  shortest_path_cuda<<<1,MAT_SIZE>>>(DevMat);
  
  
  cudasafe( hipFree(DevMat), "hipFree" );
  
  // Print path taken 
  printf("\nShortest Path: %i -> ", shortestpath);
  while(!is_empty(&result)) {
    printf("%i,", pop(&result));
  }
  printf("\n");
  
  return 0;
}

/*
 * Path is returned as 1 for right, 0 for down
 * Note: this algorithm goes from top left to bottom right corner
 * @return - int shortest path
 */
__global__ void shortest_path_cuda(int * mat) {
  
  // Copy Mat to shared memory
  __shared__ int matrix[MAT_SIZE * MAT_SIZE];
  for(int i = 0; i < MAT_SIZE-1; i++) {
    int idx = i * MAT_SIZE + threadIdx.x;
    matrix[idx] = mat[idx];
  }
  
  // Compute shortest path
  
  
  __syncthreads();
}



