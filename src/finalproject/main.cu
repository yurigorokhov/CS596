#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#define MAT_SIZE 10240
#define BLOCK_SIZE 512
#define MAX_ELEMENT 12

#include "stack.h"
#include "../include/cuda_util.h"

// Matrix to be represented as single array
typedef struct {
  int * array;
  int columns;
} Matrix;


#include "shortestpath_cuda.h"
#include "shortestpath.h"

int main(int argc, char**argv) {
  
  printf("\nSHORTEST PATH: %i x %i\n\n", MAT_SIZE, MAT_SIZE);
  
  hipEvent_t start, stop;
  float elapsedTime, elapsedTime2;
  
  // Create a matrix and populate it with random data
  Matrix mat;
  mat.array = (int*)malloc(MAT_SIZE * MAT_SIZE * sizeof(int));
  mat.columns = MAT_SIZE;
  srand ( time(NULL) );
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++) {
    mat.array[i] = rand() % MAX_ELEMENT;
  }
  
  // ######### CUDA #########
  printf("CUDA Implementation: ");
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Copy matrix to global memory
  int *DevMat, *dev_shortest_path, *dev_result_stack, *dev_calc_matrix;
  cudasafe( hipMalloc((void**)&dev_shortest_path, sizeof(int)), "hipMalloc" );
  cudasafe( hipMalloc((void**)&DevMat, MAT_SIZE * MAT_SIZE * sizeof(int)), "hipMalloc" );
  cudasafe( hipMalloc((void**)&dev_result_stack, MAT_SIZE * 2 * sizeof(int)), "hipMalloc" );
  cudasafe( hipMemcpy(DevMat, mat.array, MAT_SIZE * MAT_SIZE * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMalloc((void**)&dev_calc_matrix, MAT_SIZE * MAT_SIZE * sizeof(int)), "hipMalloc" );
  
  // Compute shortest path with cpu
  int shortestpath = 0;
  int *result_stack = (int*)malloc(MAT_SIZE * 2 * sizeof(int));
  hipEventRecord(start,0);
  dim3 threadsPerBlock(BLOCK_SIZE);
  for(int i = 1; i < MAT_SIZE; i++) {
    dim3 blocks((int)ceil((float)(i+1) / (float)threadsPerBlock.x));
    shortest_path_cuda<<<blocks,threadsPerBlock>>>(DevMat, dev_shortest_path, dev_result_stack, dev_calc_matrix, i);
  }
  for(int i = 1; i < MAT_SIZE; i++) {    
    dim3 blocks((int)ceil((float)(MAT_SIZE-i) / (float)threadsPerBlock.x));
    shortest_path_cuda_2<<<blocks,threadsPerBlock>>>(DevMat, dev_shortest_path, dev_result_stack, dev_calc_matrix, i);
  }
  if(argc > 1)
    shortest_path_cuda_3<<<1,1>>>(DevMat, dev_shortest_path, dev_result_stack, dev_calc_matrix);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  cudasafe( hipMemcpy(&shortestpath, dev_shortest_path, sizeof(int), hipMemcpyDeviceToHost) ,"hipMemcpy");
  cudasafe( hipMemcpy(result_stack, dev_result_stack, MAT_SIZE * 2 * sizeof(int), hipMemcpyDeviceToHost) ,"hipMemcpy");
  cudasafe( hipFree(DevMat), "hipFree" );
  cudasafe( hipFree(dev_result_stack), "hipFree" );
  cudasafe( hipFree(dev_shortest_path), "hipFree" );
  cudasafe( hipFree(dev_calc_matrix), "hipFree" );
  
  // Print path taken 
  printf("\nelapsed time: %f\n", elapsedTime);
  printf("\nShortest Path: %i -> ", shortestpath);
  
  if(argc > 1) {
    int i = -1; 
    while(result_stack[++i] >= 0);
    for(i--; i >= 0; i--) {
      printf("%i,", result_stack[i]);
    }
    printf("\n");
  }
  // ######### CPU Implementation #########
  printf("\n\nCPU Implementation: ");
  
  // Create a result stack
  Stack result;
  stack_init(&result, MAT_SIZE*2);
  
  // Compute shortest path with cpu
  hipEventRecord(start,0);
  shortestpath = shortest_path_cpu(&mat, &result);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime2, start, stop);
  
  // Print path taken 
  printf("\nelapsed time: %f\n", elapsedTime2);
  printf("\nShortest Path: %i -> ", shortestpath);
  if(argc > 1) {
    while(!is_empty(&result)) {
      printf("%i,", pop(&result));
    }
    printf("\n");
  }
  printf("\nSpeedup: %f\n", elapsedTime2/elapsedTime);
  
  return 0;
}