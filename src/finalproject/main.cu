#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

#define MAT_SIZE 32
#define MAX_ELEMENT 512

#include "stack.h"
#include "../include/cuda_util.h"

// Matrix to be represented as single array
typedef struct {
  int * array;
  int columns;
} Matrix;


#include "shortestpath_cuda.h"
#include "shortestpath.h"

int main() {
  
  printf("\nSHORTEST PATH: %i x %i\n\n", MAT_SIZE, MAT_SIZE);
  
  hipEvent_t start, stop;
  float elapsedTime;
  
  // Create a matrix and populate it with random data
  Matrix mat;
  mat.array = (int*)malloc(MAT_SIZE * MAT_SIZE * sizeof(int));
  mat.columns = MAT_SIZE;
  srand ( time(NULL) );
  for(int i = 0; i < MAT_SIZE * MAT_SIZE; i++) {
    mat.array[i] = rand() % MAX_ELEMENT;
  }
  
  // ######### CUDA #########
  printf("CUDA Implementation: ");
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Copy matrix to global memory
  int *DevMat, *dev_shortest_path, *dev_result_stack;
  cudasafe( hipMalloc((void**)&dev_shortest_path, sizeof(int)), "hipMalloc" );
  cudasafe( hipMalloc((void**)&DevMat, MAT_SIZE * MAT_SIZE * sizeof(int)), "hipMalloc" );
  cudasafe( hipMalloc((void**)&dev_result_stack, MAT_SIZE * 2 * sizeof(int)), "hipMalloc" );
  cudasafe( hipMemcpy(DevMat, mat.array, MAT_SIZE * MAT_SIZE * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy" );
  
  // Compute shortest path with cpu
  int shortestpath = 0;
  int *result_stack = (int*)malloc(MAT_SIZE * 2 * sizeof(int));
  hipEventRecord(start,0);
  shortest_path_cuda<<<1,MAT_SIZE>>>(DevMat, dev_shortest_path, dev_result_stack);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  cudasafe( hipMemcpy(&shortestpath, dev_shortest_path, sizeof(int), hipMemcpyDeviceToHost) ,"hipMemcpy");
  cudasafe( hipMemcpy(result_stack, dev_result_stack, MAT_SIZE * 2 * sizeof(int), hipMemcpyDeviceToHost) ,"hipMemcpy");
  cudasafe( hipFree(DevMat), "hipFree" );
  cudasafe( hipFree(dev_result_stack), "hipFree" );
  cudasafe( hipFree(dev_shortest_path), "hipFree" );
  
  // Print path taken 
  printf("\nelapsed time: %f\n", elapsedTime);
  printf("\nShortest Path: %i -> ", shortestpath);
  /*
  int i = -1; 
  while(result_stack[++i] >= 0);
  for(i--; i >= 0; i--) {
    printf("%i,", result_stack[i]);
  }
  printf("\n");
  */
  // ######### CPU Implementation #########
  printf("\n\nCPU Implementation: ");
  
  // Create a result stack
  Stack result;
  stack_init(&result, MAT_SIZE*2);
  
  // Compute shortest path with cpu
  hipEventRecord(start,0);
  shortestpath = shortest_path_cpu(&mat, &result);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  
  // Print path taken 
  printf("\nelapsed time: %f\n", elapsedTime);
  printf("\nShortest Path: %i -> ", shortestpath);
  while(!is_empty(&result)) {
    printf("%i,", pop(&result));
  }
  printf("\n");
  
  return 0;
}