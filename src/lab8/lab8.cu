#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 8 - grid configurations continued
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define PROX 48
#define SHARED_MEM_PER_BLOCK 4000
#define THREADS 2048

#ifndef GRID_Y
#define GRID_Y 4
#endif

#ifndef ARRAY_SIZE
#define ARRAY_SIZE 3 * SHARED_MEM_PER_BLOCK / 4
#endif

__global__ void sum_kernel();

int main(void) {
  hipEvent_t start, stop;
  float elapsedTime;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipEventRecord(start,0);
  
  dim3 grid(1,GRID_Y);
  sum_kernel<<<grid, THREADS / GRID_Y>>>();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\nProcessors: %i\nShared mem per block: %i", PROX, SHARED_MEM_PER_BLOCK);
  printf("\nGrid: 1x%i array of blocks, %i threads per block, S=%i -> %f\n", GRID_Y, THREADS / GRID_Y, ARRAY_SIZE, elapsedTime);
}

__global__ void sum_kernel() {
    __shared__ int filler[ARRAY_SIZE];
    filler[threadIdx.x % 16] = 0;
    int result = 0;
    for(int i = 1; i <= 1000; i++) {
      result += i;
    }
    __syncthreads();
}

