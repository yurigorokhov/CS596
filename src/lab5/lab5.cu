/**
 * Yuri Gorokhov
 * lab 5 - Modulus power of two
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define ITERATIONS 100000
#define THREADS 32
#define POW 30

__global__ void kernel_mod(int);

int main (void) {
	hipEvent_t start, stop;
	int input[POW];	
	float output[POW];
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	for(int i = 0; i < POW; i++) {
		input[i] = pow(2,i);
		hipEventRecord(start,0);
		kernel_mod<<<1,THREADS>>>(pow(2,i));
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&output[i], start, stop);
	}
	printf("[");
	for(int i = 0; i < POW; i++) {
		printf("%i, ", input[i]);
	}
	printf("\n[");
	for(int i = 0; i < POW; i++) {
		printf("%f, ", output[i]);
	}
	return 0;
}

__global__ void kernel_mod(int mod) {
	__shared__ float A[THREADS];
	int temp;
	int target = threadIdx.x % mod;
	for(int i = 1; i <= ITERATIONS; i++) {
		temp = A[target];
	}
	__syncthreads();
}

