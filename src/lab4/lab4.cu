/**
 * Yuri Gorokhov
 * lab 4 - Rows vs Columns
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 256

__global__ void kernel_row();
__global__ void kernel_col();

int main (void) {
	hipEvent_t start, stop;
	float elapsedTime;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start,0);
	kernel_row<<<1,ARRAY_SIZE>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel row time: %f\n", elapsedTime);

	hipEventRecord(start,0);
	kernel_col<<<1,ARRAY_SIZE>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel col time: %f\n", elapsedTime);
	return 0;
}

__global__ void kernel_row() {
	__shared__ float A[ARRAY_SIZE][ARRAY_SIZE];
	int sum = 0;
	for(int i = 0; i < ARRAY_SIZE-1; i++) {
		sum += A[threadIdx.x][i];
	}
	__syncthreads();
}

__global__ void kernel_col() {
	__shared__ float A[ARRAY_SIZE][ARRAY_SIZE];
	int sum = 0;
	for(int i = 0; i < ARRAY_SIZE-1; i++) {
		sum += A[i][threadIdx.x];
	}
	__syncthreads();
}

