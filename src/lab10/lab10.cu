/**
 * Yuri Gorokhov
 * lab 10 - Cuda Host Alloc
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define SIZE 10*1024*1024
#define ITERATIONS 100

float cuda_malloc_test(int size, bool up);
float cuda_host_alloc_test(int size, bool up);

int main(void) {
  float elapsedTime;
  elapsedTime = cuda_malloc_test(SIZE, true);
  printf("Time using hipMalloc (copy up): %f\n", elapsedTime);
  elapsedTime = cuda_malloc_test(SIZE, false);
  printf("Time using hipMalloc (copy down): %f\n", elapsedTime);
  
  elapsedTime = cuda_host_alloc_test(SIZE, true);
  printf("Time using hipHostAlloc (copy up): %f\n", elapsedTime);
    elapsedTime = cuda_host_alloc_test(SIZE, false);
  printf("Time using hipHostAlloc (copy down): %f\n", elapsedTime);
}

float cuda_malloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsedTime;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  a = (int*)malloc(size*sizeof(*a));
  hipMalloc((void**)&dev_a, size * sizeof(*dev_a));
  hipEventRecord(start, 0);
  for(int i = 0; i < ITERATIONS; i++) {
    if (up)
      hipMemcpy( dev_a, a, size * sizeof( *dev_a ), hipMemcpyHostToDevice );
    else
      hipMemcpy( a, dev_a, size * sizeof( *dev_a ), hipMemcpyDeviceToHost );
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  free(a);
  hipFree(dev_a);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return elapsedTime;
}

float cuda_host_alloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsedTime;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault);
  hipMalloc((void**)&dev_a, size * sizeof(*dev_a));
  hipEventRecord(start, 0);
  for(int i = 0; i < ITERATIONS; i++) {
    if (up)
      hipMemcpy( dev_a, a, size * sizeof( *dev_a ), hipMemcpyHostToDevice );
    else
      hipMemcpy( a, dev_a, size * sizeof( *dev_a ), hipMemcpyDeviceToHost );
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipHostFree(a);
  hipFree(dev_a);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return elapsedTime;
}