#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 1 - Global vs Shared memory speeds
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/cuda_util.h"

#define ARRAY_SIZE 256
#define ITERATIONS 10000000

__global__ void global_mem_kernel(int *array);
__global__ void shared_mem_kernel();

int main() {
	int *dev_array;
	hipEvent_t start, stop;
	float elapsedTime;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Global Memory
	cudasafe( hipMalloc((void**)&dev_array, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
	hipEventRecord(start,0);
	global_mem_kernel<<<1, ARRAY_SIZE>>>(dev_array);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	cudasafe( hipFree(dev_array), "hipFree" );
	printf("Global memory: %f\n", elapsedTime);

	// Shared Memory
	hipEventRecord(start,0);
	shared_mem_kernel<<<1, ARRAY_SIZE>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("Shared memory: %f\n", elapsedTime);

    return 0;
}

__global__ void global_mem_kernel(int *array) {
	for(int i = 0; i < ITERATIONS; i++) {
		int tmp = array[threadIdx.x];
		array[threadIdx.x] = tmp;
	}
}

__global__ void shared_mem_kernel() {
	int array[ARRAY_SIZE];
	for(int i = 0; i < ITERATIONS; i++) {
		int tmp = array[threadIdx.x];
		array[threadIdx.x] = tmp;
	}
}
