/**
 * Yuri Gorokhov
 * lab 3
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 512

__global__ void kernel1();
__global__ void kernel2();

int main (void) {
	hipEvent_t start, stop;
	float elapsedTime;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start,0);
	kernel1<<<1,ARRAY_SIZE>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel1 Time: %f\n", elapsedTime);

	hipEventRecord(start,0);
	kernel2<<<1,ARRAY_SIZE>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel2 Time: %f\n", elapsedTime);
	return 0;
}

__global__ void kernel1() {
	__shared__ float A[ARRAY_SIZE];
	int t = threadIdx.x;
	for(int i=1; i < ARRAY_SIZE; i *= 2) {
		__syncthreads();
		if(t%(2*i)==0)
			A[t]+=A[t+i];
	}
}

__global__ void kernel2() {
	__shared__ float A[ARRAY_SIZE];
	int t = threadIdx.x;
	for(int i = ARRAY_SIZE; i>0; i >>= 1) {
		__syncthreads();
		if(t<i)
			A[t] += A[t+i];
	}
}
