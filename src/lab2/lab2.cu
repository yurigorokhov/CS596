#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 2 - Conditional statements vs without
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/cuda_util.h"

#define ITERATIONS 10000000
#define NUM_THREADS 256

__global__ void kernel_with_conditionals();
__global__ void kernel_without_conditionals();

int main() {
	hipEvent_t start, stop;
	float elapsedTime;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// with conditionals
	hipEventRecord(start,0);
	kernel_with_conditionals<<<1, NUM_THREADS>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("Time with conditionals: %f\n", elapsedTime);

	// without conditionals
	hipEventRecord(start,0);
	kernel_without_conditionals<<<1,NUM_THREADS>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("Time without conditionals: %f\n", elapsedTime);

    return 0;
}

__global__ void kernel_with_conditionals() {
	int temp = 0;
	for(int i=0; i < ITERATIONS; i++) {
		if(threadIdx.x % 2 == 0)
			temp += 1;
		else
			temp -= 1;
	}
	__syncthreads();
}

__global__ void kernel_without_conditionals() {
	int temp = 0;
	for(int i=0; i < ITERATIONS; i++) {
		temp += (-threadIdx.x%2) + (1 - threadIdx.x%2);
	}
	__syncthreads();
}

