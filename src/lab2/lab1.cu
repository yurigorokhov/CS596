#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 2 - Conditional statements vs without
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/cuda_util.h"

#define ARRAY_SIZE 256
#define ITERATIONS 10000000

__global__ void kernel1();

int main() {
	int *dev_array;
	hipEvent_t start, stop;
	float elapsedTime;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// conditionals
	hipEventRecord(start,0);
	kernel1<<<1, ARRAY_SIZE>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("Time with conditionals: %f\n", elapsedTime);

	// without conditionals
	hipEventRecord(start,0);
	//shared_mem_kernel<<<1, ARRAY_SIZE>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("Shared memory: %f\n", elapsedTime);

    return 0;
}

__global__ void kernel1() {
	int temp = 0;
	for(int i=0; i < ITERATIONS; i++) {
		if(threadIdx.x % 2 == 0)
			temp += 1;
		else
			temp -= 1;
	}
}
