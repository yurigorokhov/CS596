#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 7 - grid configurations
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define SIZE 10000

__global__ void sum_kernel();

int main(void) {
  hipEvent_t start, stop;
  float elapsedTime;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipEventRecord(start,0);
  sum_kernel<<<1,512>>>();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\n1x1, 512 threads per block: %f", elapsedTime);
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  dim3 grid(1,2);
  hipEventRecord(start,0);
  sum_kernel<<<grid,256>>>();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\n1x2, 256 threads per block: %f", elapsedTime);
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  dim3 grid2(1,4);
  hipEventRecord(start,0);
  sum_kernel<<<grid2,128>>>();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\n1x4, 128 threads per block: %f", elapsedTime);
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  dim3 grid3(1,6);
  hipEventRecord(start,0);
  sum_kernel<<<grid3,85>>>();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\n1x8, 64 threads per block: %f", elapsedTime);
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  dim3 grid3(1,8);
  hipEventRecord(start,0);
  sum_kernel<<<grid3,64>>>();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\n1x8, 64 threads per block: %f", elapsedTime);
}

__global__ void sum_kernel() {
    int result = 0;
    for(int i = 1; i <= SIZE; i++) {
      result += i;
    }
    __syncthreads();
}

