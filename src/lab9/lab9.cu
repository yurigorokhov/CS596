#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 9 - Matrix Multiplication
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define MAT_SIZE 4

__global__ void matrix_mult(int*, int*, int);
__global__ void matrix_mult(float*, float*, int);

int main(void) {
  hipEvent_t start, stop;
  float elapsedTime;
  
  // Alocate matrices
  int *mat1, *mat2, *dev_mat1, *dev_mat2;
  mat1 = (int*)malloc(sizeof(int) * MAT_SIZE * MAT_SIZE);
  mat2 = (int*)malloc(sizeof(int) * MAT_SIZE * MAT_SIZE);
  
  // Copy matrices to device
  cudasafe( hipMalloc((void**)&dev_mat1, sizeof(int) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&dev_mat2, sizeof(int) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMemcpy(dev_mat1, mat1, sizeof(int) * MAT_SIZE * MAT_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(dev_mat2, mat2, sizeof(int) * MAT_SIZE * MAT_SIZE, hipMemcpyHostToDevice), "hipMemcpy" )
  
  // Start the kernel
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  
  
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
}

__global__ void matrix_mult(int *matrix1, int *matrix2, int size) {
}

__global__ void matrix_mult(float *matrix1, float *matrix2, int size) {
}

