#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 9 - Matrix Multiplication
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define MAT_SIZE 16
#define BLOCKS 4
#define BLOCK_SIZE MAT_SIZE/BLOCKS
#define THREADS_PER_BLOCK (MAT_SIZE/BLOCKS)*(MAT_SIZE/BLOCKS)

__global__ void matrix_mult_int(int*, int*, int*, int);
__global__ void matrix_mult_float(float*, float*, float*, int);

int main(void) {
  hipEvent_t start, stop;
  float elapsedTime;
  
  // Alocate matrices
  int *mat1, *mat2, *result, *dev_mat1, *dev_mat2, *dev_result_mat;
  mat1 = (int*)malloc(sizeof(int) * MAT_SIZE * MAT_SIZE);
  mat2 = (int*)malloc(sizeof(int) * MAT_SIZE * MAT_SIZE);
  result = (int*)malloc(sizeof(int) * MAT_SIZE * MAT_SIZE);
  
  // Copy matrices to device
  cudasafe( hipMalloc((void**)&dev_mat1, sizeof(int) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&dev_mat2, sizeof(int) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&dev_result_mat, sizeof(int) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMemcpy(dev_mat1, mat1, sizeof(int) * MAT_SIZE * MAT_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(dev_mat2, mat2, sizeof(int) * MAT_SIZE * MAT_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  
  // Start the kernel
  hipEventCreate(&start);
  hipEventCreate(&stop);
  dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  dim3 blocks(MAT_SIZE / threadsPerBlock.x, MAT_SIZE / threadsPerBlock.y);
  hipEventRecord(start,0);
  matrix_mult_int<<<blocks, threadsPerBlock>>>(dev_mat1, dev_mat2, dev_result_mat, MAT_SIZE);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time elapsed (int): %f\n", elapsedTime);
  
  // Copy the result matrix
  cudasafe( hipMemcpy(result, dev_result_mat, sizeof(int) * MAT_SIZE * MAT_SIZE, hipMemcpyDeviceToHost), "hipMemcpy" );
  cudasafe( hipFree(dev_mat1) ,"hipFree");
  cudasafe( hipFree(dev_mat2) ,"hipFree");
  cudasafe( hipFree(dev_result_mat) ,"hipFree");
  
  // Alocate matrices
  float *mat1_f, *mat2_f, *result_f, *dev_mat1_f, *dev_mat2_f, *dev_result_mat_f;
  mat1_f = (float*)malloc(sizeof(float) * MAT_SIZE * MAT_SIZE);
  mat2_f = (float*)malloc(sizeof(float) * MAT_SIZE * MAT_SIZE);
  result_f = (float*)malloc(sizeof(float) * MAT_SIZE * MAT_SIZE);
  
  // Copy matrices to device
  cudasafe( hipMalloc((void**)&dev_mat1_f, sizeof(float) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&dev_mat2_f, sizeof(float) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&dev_result_mat_f, sizeof(float) * MAT_SIZE * MAT_SIZE), "hipMalloc" );
  cudasafe( hipMemcpy(dev_mat1_f, mat1_f, sizeof(float) * MAT_SIZE * MAT_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(dev_mat2_f, mat2_f, sizeof(float) * MAT_SIZE * MAT_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  
  // Start the kernel
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  matrix_mult_float<<<blocks, threadsPerBlock>>>(dev_mat1_f, dev_mat2_f, dev_result_mat_f, MAT_SIZE);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time elapsed (float): %f\n", elapsedTime);
  
  // Copy the result matrix
  cudasafe( hipMemcpy(result_f, dev_result_mat_f, sizeof(float) * MAT_SIZE * MAT_SIZE, hipMemcpyDeviceToHost), "hipMemcpy" );
  cudasafe( hipFree(dev_mat1_f) ,"hipFree");
  cudasafe( hipFree(dev_mat2_f) ,"hipFree");
  cudasafe( hipFree(dev_result_mat_f) ,"hipFree");
}

__global__ void matrix_mult_int(int *matrix1, int *matrix2, int *result, int size) {
  int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  
  int sum = 0;
  for(int i = 0; i < size; i++) {
    sum += matrix1[row*size+i]*matrix2[col+size*i];
  }
  result[row*size + col] = sum;
  __syncthreads();
}

__global__ void matrix_mult_float(float *matrix1, float *matrix2, float *result, int size) {
  int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  
  float sum = 0;
  for(int i = 0; i < size; i++) {
    sum += matrix1[row*size+i]*matrix2[col+size*i];
  }
  result[row*size + col] = sum;
  __syncthreads();
}