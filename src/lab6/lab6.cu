#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 6 - memcpy
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define ARRAY_SIZE 10000

//__global__ void kernel_mod(int);

int main(void) {
  
  for(int i = 1; i <= 7; i++) {
    int num_arr = pow(2,i);
    int ** arrays = (int**)malloc(sizeof(int) * num_arr);
    int ** Darrays = (int**)malloc(sizeof(int) * num_arr)
    
    // allocate the arrays
    for(int k = 0; k < num_arr; k++) {
      arrays[k] = (int*)malloc(sizeof(int) * ARRAY_SIZE);
    }

    for(int k = 0; k < num_arr; k++) {
      cudasafe( hipMemcpy(Darrays[k], arrays[k], sizeof(int) * ARRAY_SIZE, "") , "hipMemcpy");
    }
    
  }
  return 0;
}

