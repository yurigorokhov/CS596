#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 6 - Integration
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define PARTITIONS 1024 // Number of partitions to divide area into, must be even
#define THREAD_PER_BLOCK 512 

float function(float);
float integrate(float (*)(float), float, float);
void error(char *);
__global__ void sum_kernel(float*, int);

int main() {
	float result;
	result = integrate(function, 0.0, 10.0);		
	printf("\nResult = %f\n\n", result);
	return 0;
}

float integrate(float (*func)(float), float lower, float upper) {
	// Create an array to store the area
	float *area_array;
	area_array = (float*)malloc(sizeof(float) * PARTITIONS);
	if(area_array == NULL)
		error("malloc failed");

	float step = (upper-lower)/PARTITIONS;
	
	// Populate the area_array by computing the Rieman Sum
	for(int i = 0; i < PARTITIONS; i++) {
		area_array[i] = func(lower + i * step) * step;
	}

	// Copy the array to the device
	float * dev_array;
	float result;
	cudasafe( hipMalloc((void**)&dev_array, sizeof(float) * PARTITIONS), "hipMalloc" );
	cudasafe( hipMemcpy(dev_array, area_array, sizeof(float) * PARTITIONS, hipMemcpyHostToDevice), "hipMemcpy" );
	free(area_array);

	// Sum up the array
	sum_kernel<<<1, PARTITIONS/2>>>(dev_array, PARTITIONS);
	cudasafe( hipMemcpy(&result, dev_array, sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy" );
	cudasafe( hipFree(dev_array), "hipFree" );
	return result;
}

float function(float x) {
	return pow(x,2);
}

void error(char * msg) {
	printf("\n%s\n\n", msg);
	exit(1);
}

__global__ void sum_kernel(float * array, int length) {
	int offset = 1;
	int thread = threadIdx.x;
	for(int d = length>>1; d > 0; d >>= 1) {
		__syncthreads();
		if(thread < d) {
			int ai = offset * (2*thread + 1) -1;
			int bi = offset * (2*thread + 2) -1;
		
			array[bi] += array[ai];
		}
		offset <<= 1;
	}

	// Copy result to beginning of array
	if(thread == 0) {
		array[0] = array[length-1];
	}
	__syncthreads();
}

