#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 6 - memcpy
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define ARRAY_SIZE 100 * 100
#define NUM_ARRAYS 8

__global__ void batched_kernel(int*);
__global__ void dummy_kernel(int*, int*, int*, int*, int*, int*, int*, int*);

int main(void) {
  
  hipEvent_t start, stop;
  float elapsedTime, elapsedTimeBatch;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate host arrays
  int *hostArray0, *hostArray1, *hostArray2, *hostArray3,
      *hostArray4, *hostArray5, *hostArray6, *hostArray7;
  int *devArray0, *devArray1, *devArray2, *devArray3,
      *devArray4, *devArray5, *devArray6, *devArray7;
      
  hostArray0 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray1 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray2 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray3 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray4 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray5 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray6 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray7 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  
  // Copying without batching
  hipEventRecord(start,0);
  
  cudasafe( hipMalloc((void**)&devArray0, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray1, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray2, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray3, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray4, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray5, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray6, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray7, sizeof(int) * ARRAY_SIZE), "hipMalloc" );

  cudasafe( hipMemcpy(devArray0, hostArray0, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray1, hostArray1, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray2, hostArray2, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray3, hostArray3, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray4, hostArray4, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray5, hostArray5, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray6, hostArray6, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray7, hostArray7, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );

  dummy_kernel<<<1,1>>>(devArray0, devArray1, devArray2, devArray3,
			devArray4, devArray5, devArray6, devArray7
 		      );
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\nTime elapsed w/out batching: %f", elapsedTime);
  
  cudasafe( hipFree(devArray0), "hipFree" );
  cudasafe( hipFree(devArray1), "hipFree" );
  cudasafe( hipFree(devArray2), "hipFree" );
  cudasafe( hipFree(devArray3), "hipFree" );
  cudasafe( hipFree(devArray4), "hipFree" );
  cudasafe( hipFree(devArray5), "hipFree" );
  cudasafe( hipFree(devArray6), "hipFree" );
  cudasafe( hipFree(devArray7), "hipFree" );
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Let's batch!
  hipEventRecord(start,0);
  int *batchedArray = (int*)malloc(ARRAY_SIZE * NUM_ARRAYS * sizeof(int));
  int *devBatchedArray;
  memcpy(batchedArray, hostArray0, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+ARRAY_SIZE, hostArray1, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+2*ARRAY_SIZE, hostArray2, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+3*ARRAY_SIZE, hostArray3, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+4*ARRAY_SIZE, hostArray4, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+5*ARRAY_SIZE, hostArray5, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+6*ARRAY_SIZE, hostArray6, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+7*ARRAY_SIZE, hostArray7, ARRAY_SIZE * sizeof(int));

  cudasafe( hipMalloc((void**)&devBatchedArray, sizeof(int) * ARRAY_SIZE * NUM_ARRAYS), "hipMalloc" );
  cudasafe( hipMemcpy(devBatchedArray, batchedArray, sizeof(int) * ARRAY_SIZE * NUM_ARRAYS, hipMemcpyHostToDevice), "hipMemcpy" );
  
  batched_kernel<<<1,1>>>(devBatchedArray);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTimeBatch, start, stop);
  printf("\nTime elapsed with batching: %f", elapsedTimeBatch);
  printf("\nSpeedup: %f\n", elapsedTimeBatch/elapsedTime * 100);
  cudasafe( hipFree(devBatchedArray), "hipFree" );
}

__global__ void dummy_kernel(int* a, int* b, int* c, int* d, int* e, int* f, int* g, int* h) 
{__syncthreads();}

__global__ void batched_kernel(int* batchArray) {
    int * devArray0 = batchArray; 
    int * devArray1 = batchArray + ARRAY_SIZE;
    int * devArray2 = batchArray + 2*ARRAY_SIZE;
    int * devArray3 = batchArray + 3*ARRAY_SIZE;
    int * devArray4 = batchArray + 4*ARRAY_SIZE;
    int * devArray5 = batchArray + 5*ARRAY_SIZE;
    int * devArray6 = batchArray + 6*ARRAY_SIZE;
    int * devArray7 = batchArray + 7*ARRAY_SIZE;
    __syncthreads();
}

