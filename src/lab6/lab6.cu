#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 6 - memcpy
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define ARRAY_SIZE 100 * 100
#define NUM_ARRAYS 16

__global__ void batched_kernel(int*);
__global__ void dummy_kernel(int*, int*, int*, int*, int*, int*, int*, int*,
			     int*, int*, int*, int*, int*, int*, int*, int* );

int main(void) {
  
  hipEvent_t start, stop;
  float elapsedTime, elapsedTimeBatch;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate host arrays
  int *hostArray0, *hostArray1, *hostArray2, *hostArray3,
      *hostArray4, *hostArray5, *hostArray6, *hostArray7,
      *hostArray8, *hostArray9, *hostArray10, *hostArray11,
      *hostArray12, *hostArray13, *hostArray14, *hostArray15;
  int *devArray0, *devArray1, *devArray2, *devArray3,
      *devArray4, *devArray5, *devArray6, *devArray7,
      *devArray8, *devArray9, *devArray10, *devArray11,
      *devArray12, *devArray13, *devArray14, *devArray15;
      
  hostArray0 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray1 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray2 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray3 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray4 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray5 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray6 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray7 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray8 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray9 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray10 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray11 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray12 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray13 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray14 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  hostArray15 = (int*)malloc(ARRAY_SIZE * sizeof(int));
  
  // Copying without batching
  hipEventRecord(start,0);
  
  cudasafe( hipMalloc((void**)&devArray0, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray1, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray2, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray3, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray4, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray5, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray6, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray7, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray8, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray9, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray10, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray11, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray12, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray13, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray14, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  cudasafe( hipMalloc((void**)&devArray15, sizeof(int) * ARRAY_SIZE), "hipMalloc" );
  
  cudasafe( hipMemcpy(devArray0, hostArray0, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray1, hostArray1, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray2, hostArray2, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray3, hostArray3, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray4, hostArray4, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray5, hostArray5, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray6, hostArray6, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray7, hostArray7, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray0, hostArray8, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray1, hostArray9, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray2, hostArray10, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray3, hostArray11, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray4, hostArray12, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray5, hostArray13, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray6, hostArray14, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  cudasafe( hipMemcpy(devArray7, hostArray15, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy" );
  
  dummy_kernel<<<1,1>>>(devArray0, devArray1, devArray2, devArray3,
			devArray4, devArray5, devArray6, devArray7,
			devArray8, devArray9, devArray10, devArray11,
			devArray12, devArray13, devArray14, devArray15
 		      );
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\nTime elapsed w/out batching: %f", elapsedTime);
  
  cudasafe( hipFree(devArray0), "hipFree" );
  cudasafe( hipFree(devArray1), "hipFree" );
  cudasafe( hipFree(devArray2), "hipFree" );
  cudasafe( hipFree(devArray3), "hipFree" );
  cudasafe( hipFree(devArray4), "hipFree" );
  cudasafe( hipFree(devArray5), "hipFree" );
  cudasafe( hipFree(devArray6), "hipFree" );
  cudasafe( hipFree(devArray7), "hipFree" );
  cudasafe( hipFree(devArray8), "hipFree" );
  cudasafe( hipFree(devArray9), "hipFree" );
  cudasafe( hipFree(devArray10), "hipFree" );
  cudasafe( hipFree(devArray11), "hipFree" );
  cudasafe( hipFree(devArray12), "hipFree" );
  cudasafe( hipFree(devArray13), "hipFree" );
  cudasafe( hipFree(devArray14), "hipFree" );
  cudasafe( hipFree(devArray15), "hipFree" );
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Let's batch!
  hipEventRecord(start,0);
  int *batchedArray = (int*)malloc(ARRAY_SIZE * NUM_ARRAYS * sizeof(int));
  int *devBatchedArray;
  memcpy(batchedArray, hostArray0, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+ARRAY_SIZE, hostArray1, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+2*ARRAY_SIZE, hostArray2, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+3*ARRAY_SIZE, hostArray3, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+4*ARRAY_SIZE, hostArray4, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+5*ARRAY_SIZE, hostArray5, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+6*ARRAY_SIZE, hostArray6, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+7*ARRAY_SIZE, hostArray7, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+8*ARRAY_SIZE, hostArray8, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+9*ARRAY_SIZE, hostArray9, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+10*ARRAY_SIZE, hostArray10, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+11*ARRAY_SIZE, hostArray11, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+12*ARRAY_SIZE, hostArray12, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+13*ARRAY_SIZE, hostArray13, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+14*ARRAY_SIZE, hostArray14, ARRAY_SIZE * sizeof(int));
  memcpy(batchedArray+15*ARRAY_SIZE, hostArray15, ARRAY_SIZE * sizeof(int));
  
  cudasafe( hipMalloc((void**)&devBatchedArray, sizeof(int) * ARRAY_SIZE * NUM_ARRAYS), "hipMalloc" );
  cudasafe( hipMemcpy(devBatchedArray, batchedArray, sizeof(int) * ARRAY_SIZE * NUM_ARRAYS, hipMemcpyHostToDevice), "hipMemcpy" );
  
  batched_kernel<<<1,1>>>(devBatchedArray);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTimeBatch, start, stop);
  printf("\nTime elapsed with batching: %f", elapsedTimeBatch);
  printf("\nSpeedup: %f\n", elapsedTimeBatch/elapsedTime * 100);
  cudasafe( hipFree(devBatchedArray), "hipFree" );
}

__global__ void dummy_kernel(int* a, int* b, int* c, int* d, int* e, int* f, int* g, int* h,
			     int* i, int* j, int* k, int* l, int* m, int* n, int* o, int* p ) 
{__syncthreads();}

__global__ void batched_kernel(int* batchArray) {
    int * devArray0 = batchArray; 
    int * devArray1 = batchArray + ARRAY_SIZE;
    int * devArray2 = batchArray + 2*ARRAY_SIZE;
    int * devArray3 = batchArray + 3*ARRAY_SIZE;
    int * devArray4 = batchArray + 4*ARRAY_SIZE;
    int * devArray5 = batchArray + 5*ARRAY_SIZE;
    int * devArray6 = batchArray + 6*ARRAY_SIZE;
    int * devArray7 = batchArray + 7*ARRAY_SIZE;
    int * devArray8 = batchArray + 8*ARRAY_SIZE;
    int * devArray9 = batchArray + 9*ARRAY_SIZE;
    int * devArray10 = batchArray + 10*ARRAY_SIZE;
    int * devArray11 = batchArray + 11*ARRAY_SIZE;
    int * devArray12 = batchArray + 12*ARRAY_SIZE;
    int * devArray13 = batchArray + 13*ARRAY_SIZE;
    int * devArray14 = batchArray + 14*ARRAY_SIZE;
    int * devArray15 = batchArray + 15*ARRAY_SIZE;
    __syncthreads();
}

