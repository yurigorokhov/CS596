#include "hip/hip_runtime.h"
/**
 * Yuri Gorokhov
 * lab 4-2 - Constant memory test
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "../include/cuda_util.h"

#define SIZE 2048
#define ITERATIONS 5000

__constant__ int array[SIZE];

__global__ void read_kernel(int);

int main(void) {
  hipEvent_t start, stop;
  float elapsedTime;
  
  // Initialize Array
  int* hostArray = (int*)malloc(SIZE * sizeof(int));
  for(int i = 0; i < SIZE; i++)
    hostArray[i] = i;
  
  // Copy Array
  cudasafe( hipMemcpyToSymbol(HIP_SYMBOL(array), hostArray, SIZE * sizeof(int), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol" );
  
  for(int n = 1; n <= 16; n++) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start,0);
  
    read_kernel<<<1,256>>>(n);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\nN = %i -> %f", n, elapsedTime);
  }
}

__global__ void read_kernel(int n) {
    int a;
    for(int i = 0; i < ITERATIONS; i++) {
      a = array[128*(threadIdx.x % n)];
    }
    __syncthreads();
}

